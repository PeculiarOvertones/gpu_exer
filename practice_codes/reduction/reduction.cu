#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>
/** Compile with one of three options for matrix multiplication:
  * NAIVE, CONSTMEM, TILED_CONSTMEM_TYPE_1, TILED_CONSTMEM_TYPE_2, TILED_CONSTMEM_CACHEHALO
  * For Printing use flag: PRINT
  **/

#define NUM_BINS 7

#ifdef NAIVE
const int NUM_BINSE = 32;
#elif REGISTERTILING_THREADCOARSENING
const int IN_TILE_SIZE = 32;
const int OUT_TILE_SIZE = IN_TILE_SIZE - 2*STENCIL_RADIUS;
#endif


sum=0;
for(int i=0; i<length; ++i) {

}


#ifdef NAIVE
__global__ void reduce_sum_naive(float *output, const float *in, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(ceil(in.size()/2), 1, 1);
      dim3 gridDim(1,1,1);
      */
    /*This ones with better control divergence and memory coalescence*/

    unsigned int i = 2*threadIdx.x;

    for(unsigned int stride = 1; stride <= blockDim.x ; stride *= 2) {
       if(threadIdx.x % stride  == 0) 
       {
	   if(i+stride < length) 
	   {
               in[i] += in[i + stride];
	   }
       }

       __syncthreads();
    }

    if(threadIdx.x == 0) 
    {
        *output = in[0];
    }
}
#endif


#ifdef CONVERGENT
__global__ void reduce_sum_convergent(float *output, const float *in, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(ceil(in.size()/2), 1, 1);
      dim3 gridDim(1,1,1);
      */
    /*less control divergence and better memory coalescing*/

    unsigned int i = threadIdx.x;

    for(unsigned int stride = blockDim.x; stride >=1 ; stride /= 2) 
    {
       if(threadIdx.x < stride) 
       {
	   if(i + stride < length) 
	   {
               in[i] += in[i + stride]; /*no need for atomic*/
	   }
       }
       __syncthreads();
    }

    if(threadIdx.x == 0) 
    {
        *output = in[0];
    }
}
#endif

#ifdef SHAREDMEM
__global__ void reduce_sum_sharedmem(float *output, const float *in, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(ceil(in.size()/2), 1, 1);
      dim3 gridDim(1,1,1);
      */
    /*fewer accesses to global memory*/

    __shared__ float in_s[BLOCK_SIZE];

    unsigned int i = threadIdx.x;

    in_s[i] = in[i] + in[i+blockDim.x];

    for(unsigned int stride = blockDim.x/2; stride >=1 ; stride /= 2) 
    {
       __syncthreads();

       if(threadIdx.x < stride) 
       {
	   if(i + stride < length) 
	   {
               in_s[i] += in_s[i + stride]; /*no need for atomic*/
	   }
       }
    }

    if(threadIdx.x == 0) 
    {
        *output = in_s[0];
    }
}
#endif


#ifdef HIERARCHICAL
__global__ void reduce_sum_hierarchical(float *output, const float *in) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(BLOCK_SIZE, 1, 1);
      dim3 gridDim(ceil(in.size()/(2*BLOCK_SIZE)),1,1); //Note that we are dividing by segment size
      */
    /*worked with large data*/

    __shared__ float in_s[BLOCK_SIZE];

    unsigned int segment =  2*blockDim.x*blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    in_s[threadIdx.x] = in[i] + in[i + blockDim.x];

    for(unsigned int stride = blockDim.x/2; stride >=1 ; stride /= 2) 
    {
       __syncthreads();

       if(threadIdx.x < stride) 
       {
           in_s[threadIdx.x] += in_s[threadIdx.x + stride]; /*no need for atomic*/
       }
    }

    if(threadIdx.x == 0) 
    {
        atomicAdd(output, in_s[0]);
    }
}
#endif

#ifdef THREADCOARSENING
__global__ void reduce_sum_threadcoarsening(float *output, const float *in) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(BLOCK_SIZE, 1, 1);
      dim3 gridDim(ceil(in.size()/(2*BLOCK_SIZE*COARSE_FACTOR)),1,1); //Note that we are dividing by segment size
      */

    __shared__ float in_s[BLOCK_SIZE];

    unsigned int segment =  COARSE_FACTOR*2*blockDim.x*blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    in_s[threadIdx.x] = 0.;
    float sum = 0.;
    for (int c=0; c < 2*COARSE_FACTOR; ++c) 
    {
        sum += in[i + c*blockDim.x];
    }
    in_s[threadIdx.x] = sum;

    for(unsigned int stride = blockDim.x/2; stride >=1 ; stride /= 2) 
    {
       __syncthreads();

       if(threadIdx.x < stride) 
       {
           in_s[threadIdx.x] += in_s[threadIdx.x + stride]; /*no need for atomic*/
       }
    }

    if(threadIdx.x == 0) 
    {
        atomicAdd(output, in_s[0]);
    }
}
#endif


void set_zero(float *M) 
{
    if(M != NULL) 
    {	
        int size = sizeof(M)/sizeof(M[0]);

	std::cout << "setting array of size: " << size << " to zero\n";
        for (int i = 0; i < size; ++i) 
        {
            M[i] = 0;    
        }
    }
}


void print_matrix(const float *M, int COL, int ROW) 
{
    for (int row = 0; row < ROW; ++row) 
    {
        for (int col = 0; col < COL; ++col) 
	{
            std::cout << std::setw(5) << M[row*COL + col];
	}
        std::cout << "\n";
    }
    std::cout << "\n";
}


void check_error(const float* h_output, const float* answer_check, const int size) {

    bool test_passed = true;
    for(int n=0; n<size; ++n) {	
        if(h_output[n] != answer_check[n]) {
           std::cout << "error: n, output, correct_ans:" << std::setw(10) << n << std::setw(10) << h_output[n] << std::setw(10) << answer_check[n] << "\n";
	   test_passed = false;
           break; 	    
        }
    }
    if(test_passed) std::cout << "Matrix Convolution Test Passed! \n";
}

int main (int argc, char* argv[])
{ 
    /*define dimensions*/ 
    /*A (Height x InnerSize)  x B (InnerSize x Width)  = M (Height x Width) **/
    const int N = 512;
    const int FilterSize = 2*FILTER_RADIUS+1;

    const int matA_memsize = Height*Width*sizeof(float);
    const int matM_memsize = Height*Width*sizeof(float);
    const int matF_memsize = FilterSize*FilterSize*sizeof(float);

#ifdef NAIVE 
    dim3 dimGrid(ceil(N/static_cast<float>(BLOCK_SIZE)), 
		 ceil(N/static_cast<float>(BLOCK_SIZE), 
		 ceil(N/static_cast<float>(BLOCK_SIZE)));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    std::cout << "cubic BLOCK_SIZE: " << std::setw(10) << BLOCK_SIZE  << "\n";

#elif REGISTERTILING_THREADCOARSENING
    dim3 dimGrid(ceil(N/static_cast<float>(OUT_TILE_SIZE)), 
	         ceil(N/static_cast<float>(OUT_TILE_SIZE)), 
		 ceil(N/static_cast<float>(OUT_TILE_SIZE));

    dim3 dimBlock(IN_TILE_SIZE, IN_TILE_SIZE, IN_TILE_SIZE);
    std::cout << "IN_TILE_SIZE, OUT_TILE_SIZE (square): " << std::setw(10) << IN_TILE_SIZE  << std::setw(10) << OUT_TILE_SIZE << "\n";
#endif

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);

    /*print cuda device properties*/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devID);
    std::cout << "\nDevice: " << prop.name << "\n";
    std::cout << "Matrix sizes (height, width, filter size): "    << std::setw(10) << Height << std::setw(10) << Width << std::setw(10) << FilterSize << "\n";
    std::cout << "dimGrid (x,y,z):  "<< std::setw(10) << dimGrid.x  << std::setw(10) << dimGrid.y << std::setw(10) << dimGrid.z << "\n";
    std::cout << "dimBlock (x,y,z): "<< std::setw(10) << dimBlock.x << std::setw(10) << dimBlock.y << std::setw(10) << dimBlock.z << "\n";

    std::cout << "\nconstant memory (KB): " << prop.totalConstMem/1024 << "\n";
    std::cout << "total global memory (GB): " << prop.totalGlobalMem/(pow(1024,3)) << "\n";
    std::cout << "shared memory per block (KB): " << prop.sharedMemPerBlock/1024 << "\n";
    std::cout << "shared memory per multiprocessor (KB): " << prop.sharedMemPerMultiprocessor/1024 << "\n";
    std::cout << "register per block: " << prop.regsPerBlock << "\n";
    std::cout << "register per multiprocessor: " << prop.regsPerMultiprocessor << "\n";
    std::cout << "multiProcessorCount: " << prop.multiProcessorCount << "\n";
    std::cout << "warpSize: " << prop.warpSize<< "\n";
    /*hipSetDevice(devID)*/

    /*define arrays on host and device*/
    /*A*B = M*/
    float* h_A = (float *) malloc(matA_memsize);
    float* h_F = (float *) malloc(matF_memsize);
    float* h_M = (float *) malloc(matM_memsize);

    float* M_check = (float *) malloc(matM_memsize);

    float* d_A = NULL;
    hipMalloc(&d_A, matA_memsize);
    float* d_F = NULL;
    hipMalloc(&d_F, matF_memsize);
    float* d_M = NULL;
    hipMalloc(&d_M, matM_memsize);

    /*initializing input array*/
    for (int j=0; j < Height; ++j) {
	for (int i=0; i < Width; ++i) {
	    h_A [j*Width + i] = static_cast<float>(j);
	}
    }
    for (int j=0; j < FilterSize; ++j) {
	for (int i=0; i < FilterSize; ++i) {
	    h_F [j*FilterSize + i] = static_cast<float>((j));
	}
    }
    /*correct answer for error checking*/
    for (int row=0; row < Height; ++row) 
    {
	for (int col=0; col < Width; ++col) 
	{
	    float sum = 0.f;
            for(int j =  0; j < FilterSize; ++j) 
	    {
                for(int i = 0; i < FilterSize; ++i) 
	        {
	            int inCol = col + i - FILTER_RADIUS; 		    
	            int inRow = row + j - FILTER_RADIUS;
	            if(inRow >= 0 && inRow < Height && inCol >=0 && inCol < Width) 
	            {
                        sum += h_A[inRow*Width + inCol] * h_F[j*FilterSize + i];     
	            }
	        }
	    }
	    M_check [row*Width + col] = sum;
	}
    }

#ifdef PRINT
    std::cout << "\nWriting A matrix:\n";
    print_matrix(h_A, Width, Height);

    std::cout << "Writing Filter F:\n";
    print_matrix(h_F, FilterSize, FilterSize);

    std::cout << "Writing correct answer for M matrix:\n";
    print_matrix(M_check, Width, Height);
#endif

    hipMemcpy(d_A, h_A, matA_memsize, hipMemcpyHostToDevice);

#ifdef NAIVE
    hipMemcpy(d_F, h_F, matF_memsize, hipMemcpyHostToDevice);
#else  //use constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(F), h_F, matF_memsize);
#endif

    hipMemset(d_M, 0, matM_memsize);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;
    hipEventRecord(startEvent, 0);

#ifdef NAIVE
    stencil_naive<<<dimGrid, dimBlock>>>(d_out, d_in, N);
#elif REGISTERTILING_THREADCOARSENING
    stencil_registertiling_threadcoarsening<<<dimGrid, dimBlock>>>(d_out, d_in, N);
#endif

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    std::cout << "\nElapsed time to run kernel (ms): " << ms << "\n";

    hipMemcpy(h_M, d_M, matM_memsize, hipMemcpyDeviceToHost); 
  
#ifdef PRINT
    std::cout << "Writing M matrix:\n";
    print_matrix(h_M, Width, Height);
#endif

   check_error(h_M, M_check, Width*Height);

//error_exit:
    /*free memory*/
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    free(h_A);
    free(h_F);
    free(h_M);
    free(M_check);

    hipFree(d_A);
    hipFree(d_F);
    hipFree(d_M);

    hipDeviceReset();
    return 0;
}

