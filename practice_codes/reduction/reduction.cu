
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>

/* Compile with one of the following options for histogram:
 * NAIVE
 * CONVERGENT
 * SHAREDMEM
 * HIERARCHICAL
 * THREADCOARSENING
 *
 * Use PRINT_INPUT for printing input data.
 **/
#ifdef THREADCOARSENING
const int COARSE_FACTOR = 2;
#endif

#ifdef NAIVE
__global__ void reduce_sum_naive(float *output, float *in) 
{
    /* - this kernel doesn't preserve input array. 
     * - Assume kernel is launched with:
     *   threadsPerBlock = dataLength/2
     *   gridDim.x = 1
     */

    unsigned int i = 2*threadIdx.x;

    for(unsigned int stride = 1; stride <= blockDim.x ; stride *= 2) {
       if(threadIdx.x % stride  == 0) {
            in[i] += in[i + stride];
       }
       __syncthreads();
    }

    if(threadIdx.x == 0) {
        *output = in[0];
    }
}
#endif


#ifdef CONVERGENT
__global__ void reduce_sum_convergent(float *output, float *in) 
{
    /* - less control divergence and better memory coalescing.
     * - doesn't preserve input array.
     * - Assume kernel is launched with:
     *   threadsPerBlock = dataLength/2
     *   gridDim.x = 1
     **/

    unsigned int i = threadIdx.x;

    for(unsigned int stride = blockDim.x; stride >=1 ; stride /= 2)
    {
       if(threadIdx.x < stride) 
       {
           in[i] += in[i + stride]; /*no need for atomic*/
       }
       __syncthreads();
    }
    if(threadIdx.x == 0) {
        *output = in[0];
    }
}
#endif


#ifdef SHAREDMEM
__global__ void reduce_sum_sharedmem(float *output, const float *in) 
{
    /* - fewer accesses to global memory.
     * - preserves input array.
     * - Assume kernel is launched with:
     *   threadsPerBlock = dataLength/2
     *   gridDim.x = 1
     **/
    extern __shared__ float sharedmem[];
    float* in_s = sharedmem;

    unsigned int t = threadIdx.x;

    //load tile and add elements blockDim.x distance away.
    in_s[t] = in[t] + in[t+blockDim.x];

    for(unsigned int stride = blockDim.x/2; stride >=1 ; stride /= 2) {
       __syncthreads();

       if(t < stride) {
           in_s[t] += in_s[t + stride]; /*no need for atomic*/
       }
    }
    if(t == 0) {
        *output = in_s[0];
    }
}
#endif


#ifdef HIERARCHICAL
__global__ void reduce_sum_hierarchical(float *output, const float *in) 
{
    /* - works for large data.
     * - preserves input array.
     * - Assume kernel is launched with:
     *   gridDim.x = dataLength/(2*threadsPerBlock));
     **/

    extern __shared__ float sharedmem[];
    float* in_s = sharedmem;

    unsigned int segment =  2*blockDim.x*blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    in_s[threadIdx.x] = in[i] + in[i + blockDim.x];

    for(unsigned int stride = blockDim.x/2; stride >=1 ; stride /= 2) 
    {
       __syncthreads();

       if(threadIdx.x < stride) 
       {
           in_s[threadIdx.x] += in_s[threadIdx.x + stride]; /*no need for atomic*/
       }
    }

    if(threadIdx.x == 0) 
    {
        atomicAdd(output, in_s[0]);
    }
}
#endif


#ifdef THREADCOARSENING
__global__ void reduce_sum_threadcoarsening(float *output, const float *in) 
{
    /* - works for large data.
     * - preserves input array.
     * - Assume kernel is launched with:
     *   gridDim.x = dataLength/(2*COARSE_FACTOR*threadsPerBlock));
     **/
    extern __shared__ float sharedmem[];
    float* in_s = sharedmem;

    unsigned int segment =  2*COARSE_FACTOR*blockDim.x*blockIdx.x;
    unsigned int i = segment + threadIdx.x;

    in_s[threadIdx.x] = 0.;
    float sum = 0.;
    for (int c=0; c < 2*COARSE_FACTOR; ++c) 
    {
        sum += in[i + c*blockDim.x];
    }
    in_s[threadIdx.x] = sum;

    for(unsigned int stride = blockDim.x/2; stride >=1 ; stride /= 2) 
    {
       __syncthreads();

       if(threadIdx.x < stride) 
       {
           in_s[threadIdx.x] += in_s[threadIdx.x + stride]; /*no need for atomic*/
       }
    }
    if(threadIdx.x == 0) 
    {
        atomicAdd(output, in_s[0]);
    }
}
#endif


void printData(const float* data, int length) {
    std::cout << "Data array: \n";
    for (int i = 0; i < length; ++i) {
        std::cout << data[i];
        if ((i + 1) % 50 == 0)
            std::cout << "\n";
        else if (i + 1 != length)
            std::cout << ", ";
    }
    std::cout << "\n";
}


/* initialize data and compute reduction sum */
void initializeDataAndComputeReduction(float* h_input, float& output, unsigned int dataLength) {
    float sum = 0.0f;

    // Seed for reproducibility (optional)
    srand(time(NULL));

    for (unsigned int i = 0; i < dataLength; ++i) {
        //h_input[i] = static_cast<float>(rand()) / RAND_MAX;  
        h_input[i] = static_cast<float>(rand() % 10);  
        sum += h_input[i];  
    }
    output = sum;
}


int main (int argc, char* argv[])
{
    /* make sure dataLength is a power of 2, 
       otherwise pad data with zeros */
    const int dataLength = 4096;

    int data_memsize = sizeof(float)*dataLength;

    std::cout << "dataLength: "   << dataLength << "\n";
    std::cout << "data size in (GB): " << data_memsize / std::pow(1024,3) << "\n";


#if defined(HIERARCHICAL) || defined(THREADCOARSENING)
    int threadsPerBlock = 64;
#else 
    int threadsPerBlock = dataLength/2;
#endif

    int blockSize = threadsPerBlock*2;
#ifdef THREADCOARSENING
    blockSize *= COARSE_FACTOR;
#endif
    int blocksPerGrid = (dataLength - 1)/blockSize + 1;

    std::cout << "\nblocks per grid: "   << blocksPerGrid << "\n";
    std::cout << "threads per block: " << threadsPerBlock << "\n";

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);

    hipError_t err = hipSetDevice(devID);
    if (err != hipSuccess) {
        std::cerr << "Failed to set device: " << hipGetErrorString(err) << "\n";
        return -1;
    }

    /*define arrays on host and device*/
    float* h_input  = (float*) malloc(data_memsize);
    float h_output = 0.;
    float output_check = 0.;

    float* d_input = NULL;
    hipMalloc(&d_input, data_memsize);

    float* d_output = NULL;
    hipMalloc(&d_output, sizeof(float));

    initializeDataAndComputeReduction(h_input, output_check, dataLength);

#ifdef PRINT_INPUT
    std::cout << "Input: \n";
    printData(h_input, dataLength);
#endif

    std::cout << "Correct Output: " << output_check << "\n";

    hipMemcpy(d_input, h_input, data_memsize, hipMemcpyHostToDevice);
    hipMemset(d_output, 0, sizeof(float));

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;
    hipEventRecord(startEvent, 0);

#ifdef NAIVE
    reduce_sum_naive<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_input);
#elif CONVERGENT
    reduce_sum_convergent<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_input);
#elif SHAREDMEM
    reduce_sum_sharedmem<<<blocksPerGrid, threadsPerBlock, threadsPerBlock>>>(d_output, d_input);
#elif HIERARCHICAL
    reduce_sum_hierarchical<<<blocksPerGrid, threadsPerBlock, threadsPerBlock>>>
                           (d_output, d_input);
#elif THREADCOARSENING
    reduce_sum_threadcoarsening<<<blocksPerGrid, threadsPerBlock, threadsPerBlock>>>
                               (d_output, d_input);
#endif

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    std::cout << "\nElapsed time to run kernel (ms): " << ms << "\n";

    //no need for device synchronize here since event synchronize is used before.
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "GPU computed Output: " << h_output << "\n";

    if(std::fabs(h_output - output_check) < 1e-8) {
        std::cout << "Reduction Test Passed!\n";
    }
    else {
    std::cout << "Reduction Test Failed! correct value: " << output_check 
              << " GPU value: " << h_output << "\n";
    }

    /*free memory*/
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    free(h_input); h_input = NULL;

    hipFree(d_input); d_input = NULL;
    hipFree(d_output); d_output = NULL;

    hipDeviceReset();
    return 0;

}
