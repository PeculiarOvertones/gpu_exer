#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>
/** Compile with one of three options for matrix multiplication:
  * NAIVE, CONSTMEM, TILED_CONSTMEM_TYPE_1, TILED_CONSTMEM_TYPE_2, TILED_CONSTMEM_CACHEHALO
  * For Printing use flag: PRINT
  **/

#define NUM_BINS 7

#ifdef NAIVE
const int NUM_BINSE = 32;
#elif REGISTERTILING_THREADCOARSENING
const int IN_TILE_SIZE = 32;
const int OUT_TILE_SIZE = IN_TILE_SIZE - 2*STENCIL_RADIUS;
#endif

#ifdef NAIVE
__global__ void hist_naive(float *hist, const float *data, unsigned int length) 
{
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
 
    if(i < length) 
    {
        int value = data[i] - 'a';

        if(value >=0 && value < 26) 
        {
            atomic(&hist[value/BIN_SIZE],1);
        }
    }
}
#endif

#ifdef PRIVATIZATION

#endif

#ifdef PRIVATIZATION_SHAREDMEM
__global__ void hist_privatization(float *hist, const float *data, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(block_size, 1, 1);
      dim3 gridDim(ceil(length/block_size),1,1);
      */

    __shared__ unsigned int hist_s[NUM_BINS];

    /*initialize shared memory*/
    for (unsigned int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x) 
    {
        hist_s[bin] = 0u;    
    }

    __syncthreads();

    /*map to shared memory histogram*/
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < length) 
    {
        int value = data[i] - 'a';

        if(value >=0 && value < 26) 
        {
            atomicAdd(&(hist_s[value/BIN_SIZE]),1);
        }
    }

    __syncthreads();


    /*commit to global memory (all blocks commit to global memory)*/
    /*Each thread is assigned to a bin. 
      Number of bins can be greater than the number of threads in a block*/
    for(int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x ) 
    {
        int binValue = hist_s[bin];
        if(binValue > 0) 
        {
            atomicAdd(&(hist[bin]), binValue);
        }
    }
}
#endif

#ifdef PRIVATIZATION_SHAREDMEM_THREADCOARSENING_CONTIGUOUS
__global__ void hist_privatization(float *hist, const float *data, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(block_size, 1, 1);
      dim3 gridDim(ceil(length/(block_size*COARSEN_FACTOR)),1,1);
      */

    __shared__ unsigned int hist_s[NUM_BINS];

    /*initialize shared memory*/
    for (unsigned int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x) 
    {
        hist_s[bin] = 0u;    
    }

    __syncthreads();

    /*map to shared memory histogram*/
    /*Note forloop*/
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;
    for(int i=tid*COARSE_FACTOR; i< min((tid+1)*COARSEN_FACTOR,length); ++i) 
    {
        int value = data[i] - 'a';
        if(value >=0 && value < 26) 
        {
            atomicAdd(&(hist_s[value/BIN_SIZE]),1);
        }
    }
    __syncthreads();

    /*commit to global memory (all blocks commit to global memory)*/
    for(int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x ) 
    {
        int binValue = hist_s[bin];
        if(binValue > 0) 
        {
            atomicAdd(&(hist[bin]), binValue);
        }
    }
}
#endif

#ifdef PRIVATIZATION_SHAREDMEM_THREADCOARSENING_INTERLEAVED
__global__ void hist_privatization(float *hist, const float *data, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(block_size, 1, 1);
      dim3 gridDim(ceil(length/(block_size*COARSEN_FACTOR)),1,1);
      */

    __shared__ unsigned int hist_s[NUM_BINS];

    /*initialize shared memory*/
    for (unsigned int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x) 
    {
        hist_s[bin] = 0u;    
    }

    __syncthreads();

    /*map to shared memory histogram*/
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;

    /*Note forloop bound and increment*/
    for(int i=tid; i<length; i+= gridDim.x*blockDim.x) 
    {
        int value = data[i] - 'a';
        if(value >=0 && value < 26) 
        {
            atomicAdd(&(hist_s[value/BIN_SIZE]),1);
        }
    }
    __syncthreads();

    /*commit to global memory (all blocks commit to global memory)*/
    for(int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x ) 
    {
        int binValue = hist_s[bin];
        if(binValue > 0) 
        {
            atomicAdd(&(hist[bin]), binValue);
        }
    }
}
#endif

#ifdef PRIVATIZATION_SHAREDMEM_THREADCOARSENING_INTERLEAVED_AGGREGATION
__global__ void hist_privatization(float *hist, const float *data, unsigned int length) 
{
    /*assume that the kernal is launched with 
      dim3 blockDim(block_size, 1, 1);
      dim3 gridDim(ceil(length/(block_size*COARSEN_FACTOR)),1,1);
      */

    __shared__ unsigned int hist_s[NUM_BINS];

    /*initialize shared memory*/
    for (unsigned int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x) 
    {
        hist_s[bin] = 0u;    
    }

    __syncthreads();
   
    unsigned int accumulator = 0;
    int prevBinIdx = -1; 

    /*map to shared memory histogram*/
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;

    /*Note forloop bound and increment*/
    for(int i=tid; i<length; i+= gridDim.x*blockDim.x) 
    {
	    
        int value = data[i] - 'a';
        if(value >=0 && value < 26) 
        {
            int bin = value/BIN_SIZE;		
	    if(bin == prevBinIdx) 
	    {
		++accumulator;
	    }
	    else 
	    {
		if(accumulator > 0) 
		{    
                   atomicAdd(&(hist_s[prevBinIdx]),accumulator);
		}
		accumulator = 1;
		prevBinIdx = bin;
	    }
        }
    }
    if(accumulator > 0) 
    {
        atomicAdd(&(hist_s[prevBinIdx]),accumulator);
    }
    __syncthreads();

    /*commit to global memory (all blocks commit to global memory)*/
    for(int bin=threadIdx.x; bin < NUM_BINS; bin+= blockDim.x ) 
    {
        int binValue = hist_s[bin];
        if(binValue > 0) 
        {
            atomicAdd(&(hist[bin]), binValue);
        }
    }
}
#endif


void set_zero(float *M) 
{
    if(M != NULL) 
    {	
        int size = sizeof(M)/sizeof(M[0]);

	std::cout << "setting array of size: " << size << " to zero\n";
        for (int i = 0; i < size; ++i) 
        {
            M[i] = 0;    
        }
    }
}


void print_matrix(const float *M, int COL, int ROW) 
{
    for (int row = 0; row < ROW; ++row) 
    {
        for (int col = 0; col < COL; ++col) 
	{
            std::cout << std::setw(5) << M[row*COL + col];
	}
        std::cout << "\n";
    }
    std::cout << "\n";
}


void check_error(const float* h_output, const float* answer_check, const int size) {

    bool test_passed = true;
    for(int n=0; n<size; ++n) {	
        if(h_output[n] != answer_check[n]) {
           std::cout << "error: n, output, correct_ans:" << std::setw(10) << n << std::setw(10) << h_output[n] << std::setw(10) << answer_check[n] << "\n";
	   test_passed = false;
           break; 	    
        }
    }
    if(test_passed) std::cout << "Matrix Convolution Test Passed! \n";
}

int main (int argc, char* argv[])
{ 
    /*define dimensions*/ 
    /*A (Height x InnerSize)  x B (InnerSize x Width)  = M (Height x Width) **/
    const int N = 512;
    const int FilterSize = 2*FILTER_RADIUS+1;

    const int matA_memsize = Height*Width*sizeof(float);
    const int matM_memsize = Height*Width*sizeof(float);
    const int matF_memsize = FilterSize*FilterSize*sizeof(float);

#ifdef NAIVE 
    dim3 dimGrid(ceil(N/static_cast<float>(BLOCK_SIZE)), 
		 ceil(N/static_cast<float>(BLOCK_SIZE), 
		 ceil(N/static_cast<float>(BLOCK_SIZE)));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    std::cout << "cubic BLOCK_SIZE: " << std::setw(10) << BLOCK_SIZE  << "\n";

#elif REGISTERTILING_THREADCOARSENING
    dim3 dimGrid(ceil(N/static_cast<float>(OUT_TILE_SIZE)), 
	         ceil(N/static_cast<float>(OUT_TILE_SIZE)), 
		 ceil(N/static_cast<float>(OUT_TILE_SIZE));

    dim3 dimBlock(IN_TILE_SIZE, IN_TILE_SIZE, IN_TILE_SIZE);
    std::cout << "IN_TILE_SIZE, OUT_TILE_SIZE (square): " << std::setw(10) << IN_TILE_SIZE  << std::setw(10) << OUT_TILE_SIZE << "\n";
#endif

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);

    /*print cuda device properties*/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devID);
    std::cout << "\nDevice: " << prop.name << "\n";
    std::cout << "Matrix sizes (height, width, filter size): "    << std::setw(10) << Height << std::setw(10) << Width << std::setw(10) << FilterSize << "\n";
    std::cout << "dimGrid (x,y,z):  "<< std::setw(10) << dimGrid.x  << std::setw(10) << dimGrid.y << std::setw(10) << dimGrid.z << "\n";
    std::cout << "dimBlock (x,y,z): "<< std::setw(10) << dimBlock.x << std::setw(10) << dimBlock.y << std::setw(10) << dimBlock.z << "\n";

    std::cout << "\nconstant memory (KB): " << prop.totalConstMem/1024 << "\n";
    std::cout << "total global memory (GB): " << prop.totalGlobalMem/(pow(1024,3)) << "\n";
    std::cout << "shared memory per block (KB): " << prop.sharedMemPerBlock/1024 << "\n";
    std::cout << "shared memory per multiprocessor (KB): " << prop.sharedMemPerMultiprocessor/1024 << "\n";
    std::cout << "register per block: " << prop.regsPerBlock << "\n";
    std::cout << "register per multiprocessor: " << prop.regsPerMultiprocessor << "\n";
    std::cout << "multiProcessorCount: " << prop.multiProcessorCount << "\n";
    std::cout << "warpSize: " << prop.warpSize<< "\n";
    /*hipSetDevice(devID)*/

    /*define arrays on host and device*/
    /*A*B = M*/
    float* h_A = (float *) malloc(matA_memsize);
    float* h_F = (float *) malloc(matF_memsize);
    float* h_M = (float *) malloc(matM_memsize);

    float* M_check = (float *) malloc(matM_memsize);

    float* d_A = NULL;
    hipMalloc(&d_A, matA_memsize);
    float* d_F = NULL;
    hipMalloc(&d_F, matF_memsize);
    float* d_M = NULL;
    hipMalloc(&d_M, matM_memsize);

    /*initializing input array*/
    for (int j=0; j < Height; ++j) {
	for (int i=0; i < Width; ++i) {
	    h_A [j*Width + i] = static_cast<float>(j);
	}
    }
    for (int j=0; j < FilterSize; ++j) {
	for (int i=0; i < FilterSize; ++i) {
	    h_F [j*FilterSize + i] = static_cast<float>((j));
	}
    }
    /*correct answer for error checking*/
    for (int row=0; row < Height; ++row) 
    {
	for (int col=0; col < Width; ++col) 
	{
	    float sum = 0.f;
            for(int j =  0; j < FilterSize; ++j) 
	    {
                for(int i = 0; i < FilterSize; ++i) 
	        {
	            int inCol = col + i - FILTER_RADIUS; 		    
	            int inRow = row + j - FILTER_RADIUS;
	            if(inRow >= 0 && inRow < Height && inCol >=0 && inCol < Width) 
	            {
                        sum += h_A[inRow*Width + inCol] * h_F[j*FilterSize + i];     
	            }
	        }
	    }
	    M_check [row*Width + col] = sum;
	}
    }

#ifdef PRINT
    std::cout << "\nWriting A matrix:\n";
    print_matrix(h_A, Width, Height);

    std::cout << "Writing Filter F:\n";
    print_matrix(h_F, FilterSize, FilterSize);

    std::cout << "Writing correct answer for M matrix:\n";
    print_matrix(M_check, Width, Height);
#endif

    hipMemcpy(d_A, h_A, matA_memsize, hipMemcpyHostToDevice);

#ifdef NAIVE
    hipMemcpy(d_F, h_F, matF_memsize, hipMemcpyHostToDevice);
#else  //use constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(F), h_F, matF_memsize);
#endif

    hipMemset(d_M, 0, matM_memsize);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;
    hipEventRecord(startEvent, 0);

#ifdef NAIVE
    stencil_naive<<<dimGrid, dimBlock>>>(d_out, d_in, N);
#elif REGISTERTILING_THREADCOARSENING
    stencil_registertiling_threadcoarsening<<<dimGrid, dimBlock>>>(d_out, d_in, N);
#endif

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    std::cout << "\nElapsed time to run kernel (ms): " << ms << "\n";

    hipMemcpy(h_M, d_M, matM_memsize, hipMemcpyDeviceToHost); 
  
#ifdef PRINT
    std::cout << "Writing M matrix:\n";
    print_matrix(h_M, Width, Height);
#endif

   check_error(h_M, M_check, Width*Height);

//error_exit:
    /*free memory*/
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    free(h_A);
    free(h_F);
    free(h_M);
    free(M_check);

    hipFree(d_A);
    hipFree(d_F);
    hipFree(d_M);

    hipDeviceReset();
    return 0;
}

