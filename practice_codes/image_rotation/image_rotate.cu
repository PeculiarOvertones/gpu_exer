
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>

const int TILE_SIZE = 32;
const int BLOCK_ROWS =8; /*launching fewer threads than the tile size in the y direction, i.e. each thread will read in more rows*/

__global__ void rotate_matrix_simple(float *output, const float *input, const int Width, const int Height) 
{
    /*Threadcoarsening in the row direction. Using a thread block with fewer threads than elements in a tile 
      is advantageous for the matrix transpose because each thread transposes four matrix elements, as a result 
      much of the index calculation cost is amortized over these elements.*/
    /*The loop iterates over the second dimension and not the first so that contiguous threads load and store contiguous data*/

    /*In this kernel, reads are coalesced, but writes are not*/


    /*first indices of the block are (blockIdx.y*TILE_SIZE, blockIdx.x*TILE_SIZE) */

    int inCol = blockIdx.x * TILE_SIZE + threadIdx.x;
    int inRow = blockIdx.y * TILE_SIZE + threadIdx.y;
    
    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) {
        /*row = inRow + j, col = inCol */
        output[inCol*Height + (inRow+j)] = input[(inRow+j)*Width + inCol];
    }

    //int inRow = blockIdx.y*blockDim*y + threadIdx.y;
    //int inCol = blockIdx.x*blockDim*x + threadIdx.x;

    ///*Index of Transposed matrix: [inCol][inRow] linearized as inCol*Height + inRow*/
    ////int tranWidth = Height;
    ////int tranRow = inCol; 
    ////int tranCol = inRow;

    ///*Index of output matrix rotated after switching column indices: [inCol][abs(inRow-Height)]*/
    //int outWidth = Height;
    //int outRow = inCol;
    //int outCol = abs(inRow - Height);

    //if(inRow < Height && inCol < Width) 
    //{
    //    output[outRow*outWidth + outCol] = input[inRow*Width + inCol];      
    //}
}


__global__ void rotate_matrix_sharedtile(float *output, const float *input, const int Width, const int Height) 
{
    __shared__ float tile[TILE_SIZE][TILE_SIZE+1];

    int inCol = blockIdx.x * TILE_SIZE + threadIdx.x;
    int inRow = blockIdx.y * TILE_SIZE + threadIdx.y;

    /*copy data to shared time*/
    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS)
    {
        tile[threadIdx.y+j][threadIdx.x] = input[(inRow+j)*Width + inCol];
    }
    __syncthreads(); /*we need this because threads write different data to output than they read from input*/

    /*here only the block is offset; this insures that the write will be contiguous*/
    //swapping columns after transpose
    int colId_td0 = blockIdx.y - (gridDim.y-1);
    inCol = std::abs(colId_td0)*TILE_SIZE + threadIdx.x; 
    inRow = blockIdx.x * TILE_SIZE + threadIdx.y;

    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) 
    {
        //swapping columns after transpose
	//we rotate along x because we didn't actually transpose tile after reading.
	int new_tx = threadIdx.x - (TILE_SIZE-1);
	int new_ty = threadIdx.y+j;

        output[(inRow+j)*Height + inCol] = tile[std::abs(new_tx)][new_ty];
    }
}


void set_zero(float *M) 
{
    if(M != NULL) 
    {	
        int size = sizeof(M)/sizeof(M[0]);

	std::cout << "setting array of size: " << size << " to zero\n";
        for (int i = 0; i < size; ++i) 
        {
            M[i] = 0;    
        }
    }
}


void print_matrix(const float *M, int COL, int ROW) 
{
    for (int row = 0; row < ROW; ++row) 
    {
        for (int col = 0; col < COL; ++col) 
	{
            std::cout << std::setw(5) << M[row*COL + col];
	}
        std::cout << "\n";
    }
    std::cout << "\n";
}


void check_error(const float* h_output, const float* answer_check, const int size) {

    bool test_passed = true;
    for(int n=0; n<size; ++n) {	
        if(h_output[n] != answer_check[n]) {
           std::cout << "error: n, output, correct_ans:" << std::setw(10) << n << std::setw(10) << h_output[n] << std::setw(10) << answer_check[n] << "\n";
	   test_passed = false;
           break; 	    
        }
    }
    if(test_passed) std::cout << "Image Rotation Test Passed! \n";
}

int main (int argc, char* argv[])
{ 
    /*define dimensions*/
    const int Width = 1024;
    const int Height = 512;

    const int mat_memsize = Width*Height*sizeof(float);

    dim3 dimGrid(ceil(Width/TILE_SIZE), ceil(Height/TILE_SIZE), 1);    

    dim3 dimBlock(TILE_SIZE, BLOCK_ROWS, 1);    

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);


    /*print cuda device properties*/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devID);
    std::cout << "\nDevice: " << prop.name << "\n";
    std::cout << "Matrix (width/cols, height/rows): "    << std::setw(10) << Width << std::setw(10) << Height << "\n";
    std::cout << "TILE_SIZE (width/cols, height/rows): " << std::setw(10) << TILE_SIZE  << std::setw(10) << BLOCK_ROWS << "\n";

    std::cout << "dimGrid (x,y,z):  "<< std::setw(10) << dimGrid.x  << std::setw(10) << dimGrid.y << std::setw(10) << dimGrid.z << "\n";
    std::cout << "dimBlock (x,y,z): "<< std::setw(10) << dimBlock.x << std::setw(10) << dimBlock.y << std::setw(10) << dimBlock.z << "\n";

    /*cudaSetDevice(devID)*/

    /*define arrays on host and device*/
    float* h_input = (float *) malloc(mat_memsize);
    float* h_output = (float *) malloc(mat_memsize);
    float* answer_check = (float *) malloc(mat_memsize);

    float* d_input = NULL;
    hipMalloc(&d_input, mat_memsize);

    float* d_output = NULL;
    hipMalloc(&d_output, mat_memsize);

    /*initializing input array*/
    for (int j=0; j < Height; ++j) {
	for (int i=0; i < Width; ++i) {
	    h_input [j*Width + i] = j*Width + i;
	}
    }
    /*correct answer for error checking*/
    /*first do transpose and store in answer_check*/
    for (int j=0; j < Height; ++j) {
	for (int i=0; i < Width; ++i) {
	    answer_check [i*Height + j] = h_input[j*Width + i];
	}
    }
    /*swap columns*/
    int newHeight = Width;
    int newWidth = Height;

    for(int j=0; j < newHeight; ++j) {
	for (int i=0; i < int(newWidth/2); ++i) {     
            float temp = answer_check[j*newWidth + i];	
            answer_check[j*newWidth+i] = answer_check[j*newWidth + (newWidth-i-1)];
            answer_check[j*newWidth + (newWidth-i-1)] = temp;
	}
    }
    //std::cout << "Writing input matrix:\n";
    //print_matrix(h_input, Width, Height);

    //std::cout << "Writing correct answer matrix:\n";
    //print_matrix(answer_check, Height, Width);


    /*check parameters*/
    if(Width % TILE_SIZE || Height % TILE_SIZE) {
        std::cout << "Width and Heigh must be a multipler of TILE_SIZE\n";
	goto error_exit;
    }
    if(TILE_SIZE % BLOCK_ROWS) {
        std::cout << "TILE_SIZE must be a multipler of BLOCK_ROWS\n";
	goto error_exit;
    }

    hipMemcpy(d_input, h_input, mat_memsize, hipMemcpyHostToDevice); 

    /*invoke a kernel*/
    //rotate_matrix_simple<<< dimGrid, dimBlock >>>(d_output, d_input, Width, Height);	
    rotate_matrix_sharedtile<<< dimGrid, dimBlock >>>(d_output, d_input, Width, Height);	

    hipMemcpy(h_output, d_output, mat_memsize, hipMemcpyDeviceToHost); 
  
    //std::cout << "Writing output matrix:\n";
    //print_matrix(h_output, Height, Width);

    check_error(h_output, answer_check, Width*Height);
error_exit:
    /*free memory*/
    free(h_input);
    free(h_output);
    free(answer_check);

    hipFree(d_input);
    hipFree(d_output);


    return 0;
}

