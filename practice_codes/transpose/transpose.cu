
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>

const int TILE_SIZE = 32;
const int BLOCK_ROWS = 8; /* We'll launch fewer threads than the tile size in the y direction, 
                             i.e. each thread will read in more rows */

#ifdef NAIVE
__global__ void transpose_simple(float *output, const float *input, const int Width, const int Height) 
{
    /* Threadcoarsening in the row direction. 
       Using a thread block with fewer threads than elements in a tile 
       is advantageous for the matrix transpose kernel, 
       because each thread transposes four matrix elements, 
       as a result much of the index calculation cost is amortized over these elements.
     */

    int inCol = blockIdx.x * TILE_SIZE + threadIdx.x;
    int inRow = blockIdx.y * TILE_SIZE + threadIdx.y;
    
    /* The loop iterates over the row-wise dimension (y) and not the column-wise, 
       so that contiguous threads load and store contiguous data. 
     * In this kernel, reads are coalesced, but writes are not.
     */
    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) 
    {
        int row_glo = inRow + j;
        output[inCol*Height + row_glo] = input[row_glo*Width + inCol];
    }
}
#endif

#ifdef CORNER_TURNING
__global__ void transpose_sharedtile(float *output, 
                                     const float *input, const int Width, const int Height) 
{
    __shared__ float tile[TILE_SIZE][TILE_SIZE];
	
    int inCol = blockIdx.x * TILE_SIZE + threadIdx.x;
    int inRow = blockIdx.y * TILE_SIZE + threadIdx.y;

    /* load tile */
    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) 
    {
        tile[threadIdx.y+j][threadIdx.x] = input[(inRow+j)*Width + inCol];
    }
    __syncthreads(); /* we need this because threads write different data 
                        to output than they read from input */

    /* here only the block is offset; this insures that the write will be contiguous */

    inCol = blockIdx.y * TILE_SIZE + threadIdx.x; 
    inRow = blockIdx.x * TILE_SIZE + threadIdx.y;

    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) 
    {
        output[(inRow+j)*Height + inCol] = tile[threadIdx.x][threadIdx.y+j];
    }
}
#endif


#ifdef NO_BANK_CONFLICT
__global__ void transpose_sharedtile_bankconflictavoid(float *output, 
                                                       const float *input, 
                                                       const int Width, const int Height) 
{
    /* By padding the shared memory tile width in the x direction by 1, 
       we avoid bank conflict.
       This is because at the time of writing to output, 
       all threads are accessing elements from the same column of the tile.
       In this case, if the tile width size is the same as the number of channels/2 
       (accounting for double data rate access), 
       then all threads would try to access from the same bank.

     */	
    __shared__ float tile[TILE_SIZE][TILE_SIZE+1];
	
    /* The rest of the code remains the same. */

    int inCol = blockIdx.x * TILE_SIZE + threadIdx.x;
    int inRow = blockIdx.y * TILE_SIZE + threadIdx.y;

    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) 
    {
        tile[threadIdx.y+j][threadIdx.x] = input[(inRow+j)*Width + inCol];
    }
    __syncthreads(); 

    inCol = blockIdx.y * TILE_SIZE + threadIdx.x; 
    inRow = blockIdx.x * TILE_SIZE + threadIdx.y;

    for (int j=0; j < TILE_SIZE; j += BLOCK_ROWS) 
    {
        output[(inRow+j)*Height + inCol] = tile[threadIdx.x][threadIdx.y+j];
    }
}
#endif 

void set_zero(float *M) 
{
    if(M != NULL) 
    {	
        int size = sizeof(M)/sizeof(M[0]);

	std::cout << "setting array of size: " << size << " to zero\n";
        for (int i = 0; i < size; ++i) 
        {
            M[i] = 0;    
        }
    }
}


void print_matrix(const float *M, int COL, int ROW) 
{
    for (int row = 0; row < ROW; ++row) 
    {
        for (int col = 0; col < COL; ++col) 
	{
            std::cout << std::setw(5) << M[row*COL + col];
	}
        std::cout << "\n";
    }
    std::cout << "\n";
}


void check_transpose_error(const float* h_output, const float* answer_check, const int size) {

    bool test_passed = true;
    for(int n=0; n<size; ++n) {	
        if(h_output[n] != answer_check[n]) {
           std::cout << "error: n, output, correct_ans:" << std::setw(10) << n << std::setw(10) << h_output[n] << std::setw(10) << answer_check[n] << "\n";
	   test_passed = false;
           break; 	    
        }
    }
    if(test_passed) std::cout << "Transpose Test Passed! \n";
}


int main (int argc, char* argv[])
{ 
    /*define dimensions*/
    const int Width = 1024;
    const int Height = 768;

    const int mat_memsize = Width*Height*sizeof(float);

    dim3 dimGrid(ceil(Width/TILE_SIZE), ceil(Height/TILE_SIZE), 1);    
    dim3 dimBlock(TILE_SIZE, BLOCK_ROWS, 1);    

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);

    /*print cuda device properties*/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devID);

    std::cout << "\nDevice: " << prop.name << "\n";

    std::cout << "Matrix (width/cols, height/rows): " 
        << std::setw(10) << Width << std::setw(10) << Height << "\n";

    std::cout << "TILE_SIZE (width/cols, height/rows): " 
        << std::setw(10) << TILE_SIZE  << std::setw(10) << BLOCK_ROWS << "\n";

    std::cout << "dimGrid (x,y,z):  " << std::setw(10) 
              << dimGrid.x  << std::setw(10) 
              << dimGrid.y << std::setw(10) 
              << dimGrid.z << "\n";

    std::cout << "dimBlock (x,y,z): " << std::setw(10) 
              << dimBlock.x << std::setw(10) 
              << dimBlock.y << std::setw(10) 
              << dimBlock.z << "\n";

    /*define arrays on host and device*/
    float* h_input = (float *) malloc(mat_memsize);
    float* h_output = (float *) malloc(mat_memsize);
    float* answer_check = (float *) malloc(mat_memsize);

    float* d_input = NULL;
    hipMalloc(&d_input, mat_memsize);

    float* d_output = NULL;
    hipMalloc(&d_output, mat_memsize);

    /*check parameters*/
    if(Width % TILE_SIZE || Height % TILE_SIZE) {
        std::cout << "Width and Heigh must be a multipler of TILE_SIZE\n";
	goto error_exit;
    }
    if(TILE_SIZE % BLOCK_ROWS) {
        std::cout << "TILE_SIZE must be a multipler of BLOCK_ROWS\n";
	goto error_exit;
    }

    /*initializing input array*/
    for (int j=0; j < Height; ++j) {
	    for (int i=0; i < Width; ++i) {
	        h_input [j*Width + i] = j*Width + i;
	    }
    }
    /*correct answer for error checking*/
    for (int j=0; j < Height; ++j) {
	    for (int i=0; i < Width; ++i) {
	        answer_check [i*Height + j] = h_input[j*Width + i];
	    }
    }
    #ifdef PRINT
    std::cout << "Writing input matrix:\n";
    print_matrix(h_input, Width, Height);

    std::cout << "Writing correct answer matrix:\n";
    print_matrix(answer_check, Height, Width);
    #endif

    hipMemcpy(d_input, h_input, mat_memsize, hipMemcpyHostToDevice); 

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;
    hipEventRecord(startEvent, 0);

    /*invoke a kernel*/
    #ifdef NAIVE
    transpose_simple<<< dimGrid, dimBlock >>>(d_output, d_input, Width, Height);
    #elif CORNER_TURNING
    transpose_sharedtile<<< dimGrid, dimBlock >>>(d_output, d_input, Width, Height);
    #elif NO_BANK_CONFLICT
    transpose_sharedtile_bankconflictavoid<<< dimGrid, dimBlock >>>
        (d_output, d_input, Width, Height);
    #endif

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    std::cout << "Time elapsed: " << ms << "\n";

    hipMemcpy(h_output, d_output, mat_memsize, hipMemcpyDeviceToHost); 
  
    #ifdef PRINT
    std::cout << "Writing output matrix:\n";
    print_matrix(h_output, Height, Width);
    #endif

    check_transpose_error(h_output, answer_check, Width*Height);

error_exit:
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    /*free memory*/
    free(h_input);
    free(h_output);
    free(answer_check);

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}

