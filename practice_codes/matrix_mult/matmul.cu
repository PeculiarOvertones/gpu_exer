#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>
/** Compile with one of three options for matrix multiplication:
  * NAIVE, TILED, or TILED_COARSENED
  * For Printing use flag: PRINT
  * e.g. nvcc -DTILED -DPRINT matmul.cu -o solver.x
  **/

#ifdef NAIVE
const int BLOCK_ROWS = 8; 
#elif defined(TILED) || defined(TILED_COARSENED)
const int TILE_SIZE = 32;
#endif

#ifdef TILED_COARSENED
const int COARSE_FACTOR = 4;
#endif


#ifdef NAIVE
__global__ void matmul_naive(float *M, const float *A, const float *B, const int Height, const int Width, const int InnerSize) 
{
    /*                   col 
                       [.|.|.] B (I x W)
                       [.|.|.]     
                       [.|.|.]
                       [.|.|.]
                                          
            [.|.|.|.]  [.|.|.]   
       row  [.|.|.|.]  [.|.|.]
            [.|.|.|.]  [.|.|.]
	    A (H x I)  M (H x W)         	     
       */

    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
 
    if(row < Height && col < Width) 
    {
	float sum = 0.f;    
        for(int i = 0; i < InnerSize; ++i) 
	{
	    sum += A[row*InnerSize + i] * B[i*Width + col];     	
	}
        M[row*Width+col] = sum;
    }
}
#endif


#ifdef TILED
__global__ void matmul_tiled(float *M, const float *A, const float *B, const int Height, const int Width, const int InnerSize) 
{
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int col = blockIdx.x*TILE_SIZE + threadIdx.x;
    int row = blockIdx.y*TILE_SIZE + threadIdx.y;
  
    float sum = 0.f;

    for(int istart=0; istart < InnerSize; istart += TILE_SIZE) 
    {
        /*load tile A*/
        int colA = (istart + threadIdx.x);

        if(row < Height && colA < InnerSize) 
            tile_A[threadIdx.y][threadIdx.x] = A[row*InnerSize + colA];
        else 
            tile_A[threadIdx.y][threadIdx.x] = 0.f;

        /*load tile B*/
        int rowB = (istart + threadIdx.y);

        if(rowB < InnerSize && col < Width) 
            tile_B[threadIdx.y][threadIdx.x] = B[rowB*Width + col];
        else 
            tile_B[threadIdx.y][threadIdx.x] = 0.f;

        __syncthreads();

        /*do computations*/
        for (int t=0; t<TILE_SIZE; ++t) 
        {
    	    sum += tile_A[threadIdx.y][t] * tile_B[t][threadIdx.x];    
        }

        __syncthreads();
    }

    if(row < Height && col < Width) 
    {
        M[row*Width+col] = sum;
    }

    /*Note: load individual tiles and only check for boundary conditions of respective rows and cols*/
    /*Note: Another way*/
    //  for(int phase=0; phase < (InnerSize/TILE_SIZE); ++phase) 
    //  {
    //    //In this case
    //    int colA = (phase*TILE_SIZE + threadIdx.x);
    //    int rowB = (phase*TILE_SIZE + threadIdx.y);
}
#endif


#ifdef TILED_COARSENED
__global__ void matmul_tiled_coarsened(float *M, const float *A, const float *B, const int Height, const int Width, const int InnerSize) 
{
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int col = blockIdx.x*TILE_SIZE*COARSE_FACTOR + threadIdx.x;
    int row = blockIdx.y*TILE_SIZE + threadIdx.y;
  
    float sum[COARSE_FACTOR];
    for(int c=0; c<COARSE_FACTOR; ++c) sum[c] = 0.f;

    for(int istart=0; istart < InnerSize; istart += TILE_SIZE) 
    {
        /*load tile A*/
        int colA = (istart + threadIdx.x);

        if(row < Height && colA < InnerSize) 
            tile_A[threadIdx.y][threadIdx.x] = A[row*InnerSize + colA];
        else 
            tile_A[threadIdx.y][threadIdx.x] = 0.f;
        

        int rowB = (istart + threadIdx.y);
        for(int c=0; c<COARSE_FACTOR; ++c) 
	{
            /*load tile B*/
            int colB = c*TILE_SIZE + col;

            if(rowB < InnerSize && colB < Width) 
                tile_B[threadIdx.y][threadIdx.x] = B[rowB*Width + colB];
            else 
                tile_B[threadIdx.y][threadIdx.x] = 0.f;

            __syncthreads();

            /*do computations*/
            for (int t=0; t<TILE_SIZE; ++t) 
            {
    	        sum[c] += tile_A[threadIdx.y][t] * tile_B[t][threadIdx.x];    
            }

            __syncthreads();
	}
    }

    if(row < Height && col < Width) 
    {
        for(int c=0; c<COARSE_FACTOR; ++c) M[row*Width + (c*TILE_SIZE + col)] = sum[c];
    }
}
#endif

void set_zero(float *M) 
{
    if(M != NULL) 
    {	
        int size = sizeof(M)/sizeof(M[0]);

	std::cout << "setting array of size: " << size << " to zero\n";
        for (int i = 0; i < size; ++i) 
        {
            M[i] = 0;    
        }
    }
}


void print_matrix(const float *M, int COL, int ROW) 
{
    for (int row = 0; row < ROW; ++row) 
    {
        for (int col = 0; col < COL; ++col) 
	{
            std::cout << std::setw(5) << M[row*COL + col];
	}
        std::cout << "\n";
    }
    std::cout << "\n";
}


void check_error(const float* h_output, const float* answer_check, const int size) {

    bool test_passed = true;
    for(int n=0; n<size; ++n) {	
        if(h_output[n] != answer_check[n]) {
           std::cout << "error: n, output, correct_ans:" << std::setw(10) << n << std::setw(10) << h_output[n] << std::setw(10) << answer_check[n] << "\n";
	   test_passed = false;
           break; 	    
        }
    }
    if(test_passed) std::cout << "Matrix Multiplication Test Passed! \n";
}

int main (int argc, char* argv[])
{ 
    /*define dimensions*/ 
    /*A (Height x InnerSize)  x B (InnerSize x Width)  = M (Height x Width) **/
    const int Height = 1024;
    const int InnerSize = 512;
    const int Width = 1024;

    const int matA_memsize = Height*InnerSize*sizeof(float);
    const int matB_memsize = InnerSize*Width*sizeof(float);
    const int matM_memsize = Height*Width*sizeof(float);

#ifdef NAIVE
    /*Note: ceil(Height/static_cast<float>(BLOC_ROWS)) = (Height-1)/BLOCK_ROWS + 1  if Height and TILE_SIZE are integers*/

    dim3 dimGrid(1, (Height-1)/BLOCK_ROWS + 1, 1);
    dim3 dimBlock(Width, BLOCK_ROWS, 1);
#elif TILED
    dim3 dimGrid((Width-1)/TILE_SIZE + 1, (Height-1)/TILE_SIZE + 1, 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
#elif TILED_COARSENED
    /*Note loading fewer threads in a block than the elements they will be responsible to process*/
    dim3 dimGrid((Width-1)/(TILE_SIZE*COARSE_FACTOR) + 1, (Height-1)/TILE_SIZE + 1, 1);
    //dim3 dimGrid(ceil(Width/static_cast<float>(TILE_SIZE*COARSE_FACTOR)), ceil(Height/static_cast<float>(TILE_SIZE)), 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1); 
#endif

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);

    /*print cuda device properties*/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devID);
    std::cout << "\nDevice: " << prop.name << "\n";
    std::cout << "Matrix sizes (height, innersize, width): "    << std::setw(10) << Height << std::setw(10) << InnerSize << std::setw(10) << Width << "\n";
#if defined(TILED) || defined(TILED_COARSENED)
    std::cout << "TILE_SIZE (height, width): " << std::setw(10) << TILE_SIZE  << std::setw(10) << TILE_SIZE << "\n";
#endif
    std::cout << "dimGrid (x,y,z):  "<< std::setw(10) << dimGrid.x  << std::setw(10) << dimGrid.y << std::setw(10) << dimGrid.z << "\n";
    std::cout << "dimBlock (x,y,z): "<< std::setw(10) << dimBlock.x << std::setw(10) << dimBlock.y << std::setw(10) << dimBlock.z << "\n";

    /*hipSetDevice(devID)*/

    /*define arrays on host and device*/
    /*A*B = M*/
    float* h_A = (float *) malloc(matA_memsize);
    float* h_B = (float *) malloc(matB_memsize);
    float* h_M = (float *) malloc(matM_memsize);

    float* M_check = (float *) malloc(matM_memsize);

    float* d_A = NULL;
    hipMalloc(&d_A, matA_memsize);
    float* d_B = NULL;
    hipMalloc(&d_B, matB_memsize);
    float* d_M = NULL;
    hipMalloc(&d_M, matM_memsize);

    /*initializing input array*/
    for (int j=0; j < Height; ++j) {
	for (int i=0; i < InnerSize; ++i) {
	    h_A [j*InnerSize + i] = static_cast<float>(j);
	}
    }
    for (int j=0; j < InnerSize; ++j) {
	for (int i=0; i < Width; ++i) {
	    h_B [j*Width + i] = static_cast<float>(j);
	}
    }
    /*correct answer for error checking*/
    for (int row=0; row < Height; ++row) {
	for (int col=0; col < Width; ++col) {
	    float sum = 0.f;
	    for (int i=0; i < InnerSize; ++i) {
	        sum += h_A[row*InnerSize + i] * h_B[i*Width + col];
	    }
	    M_check [row*Width + col] = sum;
	}
    }

#ifdef PRINT
    std::cout << "Writing A matrix:\n";
    print_matrix(h_A, InnerSize, Height);

    std::cout << "Writing B matrix:\n";
    print_matrix(h_B, Width, InnerSize);

    std::cout << "Writing correct answer for M matrix:\n";
    print_matrix(M_check, Width, Height);
#endif


    ///*check parameters*/
    //if(Width % TILE_SIZE || Height % TILE_SIZE) {
    //    std::cout << "Width and Heigh must be a multipler of TILE_SIZE\n";
    //    goto error_exit;
    //}
    //if(TILE_SIZE % BLOCK_ROWS) {
    //    std::cout << "TILE_SIZE must be a multipler of BLOCK_ROWS\n";
    //    goto error_exit;
    //}

    hipMemcpy(d_A, h_A, matA_memsize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matB_memsize, hipMemcpyHostToDevice);

//    hipMemset(d_M, 0, matM_memsize);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;
    hipEventRecord(startEvent, 0);

#ifdef NAIVE
    matmul_naive<<<dimGrid, dimBlock>>>(d_M, d_A, d_B, Height, Width, InnerSize);
#elif TILED
    matmul_tiled<<<dimGrid, dimBlock>>>(d_M, d_A, d_B, Height, Width, InnerSize);
#elif TILED_COARSENED
    matmul_tiled_coarsened<<<dimGrid, dimBlock>>>(d_M, d_A, d_B, Height, Width, InnerSize);
#endif

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    std::cout << "Elapsed time to run kernel (ms): " << ms << "\n";

    hipMemcpy(h_M, d_M, matM_memsize, hipMemcpyDeviceToHost); 
  
#ifdef PRINT
    std::cout << "Writing M matrix:\n";
    print_matrix(h_M, Width, Height);
#endif

   check_error(h_M, M_check, Width*Height);

//error_exit:
    /*free memory*/
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    free(h_A);
    free(h_B);
    free(h_M);
    free(M_check);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_M);

    hipDeviceReset();
    return 0;
}

