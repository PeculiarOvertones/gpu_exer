#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <math.h>
#include <iostream>
#include <assert.h>
/** Compile with one of three options for matrix multiplication:
  * NAIVE, CONSTMEM, TILED_CONSTMEM_TYPE_1, TILED_CONSTMEM_TYPE_2, TILED_CONSTMEM_CACHEHALO_TYPE_1
  * For Printing use flag: PRINT
  **/

#define FILTER_RADIUS 1

#ifndef NAIVE
__constant__ float F[2*FILTER_RADIUS+1][2*FILTER_RADIUS+1];
#endif

#ifdef TILED_CONSTMEM_TYPE_1
const int IN_TILE_SIZE = 32;
const int OUT_TILE_SIZE = IN_TILE_SIZE - 2*FILTER_RADIUS;
#endif

#ifdef NAIVE
__global__ void convolution_naive(float *M, const float *A, const float *Filter, const int Height, const int Width) 
{
    /*A = pointer to input array 
     *M = pointer to output array
     *F = pointer to filter array
     *Width and Height of input and output arrays
     *R = radius of square filter
     */ 
    
    int FilterSize = 2*FILTER_RADIUS + 1;	
    int outCol = blockIdx.x*blockDim.x + threadIdx.x;
    int outRow = blockIdx.y*blockDim.y + threadIdx.y;
 
    float sum = 0.f;    
    for(int j =  0; j < FilterSize; j++) 
    {
        for(int i = 0; i < FilterSize; i++) 
        {
    	    int inCol = outCol + i - FILTER_RADIUS;    
    	    int inRow = outRow + j - FILTER_RADIUS;    
                        	
    	    if(inRow >= 0 && inRow < Height && inCol >=0 && inCol < Width) 
    	    {
                    sum += A[inRow*Width + inCol] * Filter[j*FilterSize + i];     
    	    }
        }
    }

    if(outRow < Height && outCol < Width) 
    {
        M[outRow*Width + outCol] = sum;
    }
}
#endif

#ifdef CONSTMEM
__global__ void convolution_constmem(float *M, const float *A, const int Height, const int Width) 
{
	
    int FilterSize = 2*FILTER_RADIUS + 1;	
    int outCol = blockIdx.x*blockDim.x + threadIdx.x;
    int outRow = blockIdx.y*blockDim.y + threadIdx.y;

    float sum = 0.f;    
    for(int j =  0; j < FilterSize; j++) 
    {
        for(int i = 0; i < FilterSize; i++) 
        {
    	    int inCol = outCol + i - FILTER_RADIUS;    
    	    int inRow = outRow + j - FILTER_RADIUS;    
                        	
    	    if(inRow >= 0 && inRow < Height && inCol >=0 && inCol < Width) 
    	    {
                sum += A[inRow*Width + inCol] * F[j][i];     
    	    }
        }
    }

    if(outRow < Height && outCol < Width) 
    {
        M[outRow*Width + outCol] = sum;
    }
}
#endif


#ifdef TILED_CONSTMEM_TYPE_1

__global__ void convolution_constmem_tiled_type1(float *M, const float *A, const int Height, const int Width) 
{
	
    int FilterSize = 2*FILTER_RADIUS + 1;

    int inCol = blockIdx.x*OUT_TILE_SIZE - FILTER_RADIUS + threadIdx.x;
    int inRow = blockIdx.y*OUT_TILE_SIZE - FILTER_RADIUS + threadIdx.y;

    __shared__ tile_A[IN_TILE_SIZE][IN_TILE_SIZE];

    /*load tile*/
    if(inRow >=0 && inRow < Height && inCol >=0 && inCol <= Width) 
        tile_A[threadIdx.y][threadIdx.x] = A[inRow*Width + inCol];
    else
        tile_A[threadIdx.y][threadIdx.x] = 0.;
     
    __syncthreads();

    int local_out_colId = threadIdx.y - FILTER_RADIUS;
    int local_out_rowId = threadIdx.x - FILTER_RADIUS;

    if(local_out_colId >=0 && local_out_colId  < OUT_TILE_SIZE &&
       local_out_rowId >=0 && local_out_rowId < OUT_TILE_SIZE )
    {
        float sum = 0.f;	    
        for(int j=0; j<FilterSize; ++j) 
        {
            for(int i=0; i<FilterSize; ++i) 
            {
                sum += tile_A[local_out_rowId][local_out_colId] * F[j][i];	    
            }
        }
        M[inRow*Width + inCol] = sum;
    }

    __syncthreads();
}
#endif


#ifdef TILED_CONSTMEM_CACHEHALO_TYPE_1
//__global__ void matmul_tiled_coarsened(float *M, const float *A, const float *B, const int Height, const int Width, const int InnerSize) 
//{
//    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
//    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];
//
//    int col = blockIdx.x*TILE_SIZE*COARSE_FACTOR + threadIdx.x;
//    int row = blockIdx.y*TILE_SIZE + threadIdx.y;
//  
//    float sum[COARSE_FACTOR];
//    for(int c=0; c<COARSE_FACTOR; ++c) sum[c] = 0.f;
//
//    for(int istart=0; istart < InnerSize; istart += TILE_SIZE) 
//    {
//        /*load tile A*/
//        int colA = (istart + threadIdx.x);
//
//        if(row < Height && colA < InnerSize) 
//            tile_A[threadIdx.y][threadIdx.x] = A[row*InnerSize + colA];
//        else 
//            tile_A[threadIdx.y][threadIdx.x] = 0.f;
//        
//
//        int rowB = (istart + threadIdx.y);
//        for(int c=0; c<COARSE_FACTOR; ++c) 
//	{
//            /*load tile B*/
//            int colB = c*TILE_SIZE + col;
//
//            if(rowB < InnerSize && colB < Width) 
//                tile_B[threadIdx.y][threadIdx.x] = B[rowB*Width + colB];
//            else 
//                tile_B[threadIdx.y][threadIdx.x] = 0.f;
//
//            __syncthreads();
//
//            /*do computations*/
//            for (int t=0; t<TILE_SIZE; ++t) 
//            {
//    	        sum[c] += tile_A[threadIdx.y][t] * tile_B[t][threadIdx.x];    
//            }
//
//            __syncthreads();
//	}
//    }
//
//    if(row < Height && col < Width) 
//    {
//        for(int c=0; c<COARSE_FACTOR; ++c) M[row*Width + (c*TILE_SIZE + col)] = sum[c];
//    }
//}
#endif

void set_zero(float *M) 
{
    if(M != NULL) 
    {	
        int size = sizeof(M)/sizeof(M[0]);

	std::cout << "setting array of size: " << size << " to zero\n";
        for (int i = 0; i < size; ++i) 
        {
            M[i] = 0;    
        }
    }
}


void print_matrix(const float *M, int COL, int ROW) 
{
    for (int row = 0; row < ROW; ++row) 
    {
        for (int col = 0; col < COL; ++col) 
	{
            std::cout << std::setw(5) << M[row*COL + col];
	}
        std::cout << "\n";
    }
    std::cout << "\n";
}


void check_error(const float* h_output, const float* answer_check, const int size) {

    bool test_passed = true;
    for(int n=0; n<size; ++n) {	
        if(h_output[n] != answer_check[n]) {
           std::cout << "error: n, output, correct_ans:" << std::setw(10) << n << std::setw(10) << h_output[n] << std::setw(10) << answer_check[n] << "\n";
	   test_passed = false;
           break; 	    
        }
    }
    if(test_passed) std::cout << "Matrix Convolution Test Passed! \n";
}

int main (int argc, char* argv[])
{ 
    /*define dimensions*/ 
    /*A (Height x InnerSize)  x B (InnerSize x Width)  = M (Height x Width) **/
    const int Height = 32;
    const int Width = 32;
    const int FilterSize = 2*FILTER_RADIUS+1;

    const int matA_memsize = Height*Width*sizeof(float);
    const int matM_memsize = Height*Width*sizeof(float);
    const int matF_memsize = FilterSize*FilterSize*sizeof(float);

#if defined(NAIVE) || defined(CONSTMEM)
    dim3 dimGrid(ceil(Width/static_cast<float>(BLOCK_SIZE)), ceil(Height/static_cast<float>(BLOCK_SIZE)), 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    std::cout << "BLOCK_SIZE (height, width): " << std::setw(10) << BLOCK_SIZE  << "\n";
#elif TILED_CONSTMEM_TYPE_1
    dim3 dimGrid(ceil(Width/static_cast<float>(IN_TILE_SIZE)), ceil(Height/static_cast<float>(IN_TILE_SIZE)), 1);
    dim3 dimBlock(IN_TILE_SIZE, IN_TILE_SIZE, 1);
    std::cout << "IN_TILE_SIZE, OUT_TILE_SIZE: " << std::setw(10) << IN_TILE_SIZE  << std::setw(10) << OUT_TILE_SIZE << "\n";
#elif TILED_CONSTMEM_CACHEHALO_TYPE_1
    dim3 dimBlock(IN_TILE_SIZE, IN_TILE_SIZE, 1); 
#endif

    int devID=0;
    if(argc > 1) devID = atoi(argv[1]);

    /*print cuda device properties*/
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devID);
    std::cout << "\nDevice: " << prop.name << "\n";
    std::cout << "Matrix sizes (height, width, filter size): "    << std::setw(10) << Height << std::setw(10) << Width << std::setw(10) << FilterSize << "\n";
    std::cout << "dimGrid (x,y,z):  "<< std::setw(10) << dimGrid.x  << std::setw(10) << dimGrid.y << std::setw(10) << dimGrid.z << "\n";
    std::cout << "dimBlock (x,y,z): "<< std::setw(10) << dimBlock.x << std::setw(10) << dimBlock.y << std::setw(10) << dimBlock.z << "\n";

    std::cout << "\nconstant memory (KB): " << prop.totalConstMem/1024 << "\n";
    std::cout << "total global memory (GB): " << prop.totalGlobalMem/(pow(1024,3)) << "\n";
    std::cout << "shared memory per block (KB): " << prop.sharedMemPerBlock/1024 << "\n";
    std::cout << "shared memory per multiprocessor (KB): " << prop.sharedMemPerMultiprocessor/1024 << "\n";
    std::cout << "register per block: " << prop.regsPerBlock << "\n";
    std::cout << "register per multiprocessor: " << prop.regsPerMultiprocessor << "\n";
    std::cout << "multiProcessorCount: " << prop.multiProcessorCount << "\n";
    std::cout << "warpSize: " << prop.warpSize<< "\n";
    /*hipSetDevice(devID)*/

    /*define arrays on host and device*/
    /*A*B = M*/
    float* h_A = (float *) malloc(matA_memsize);
    float* h_F = (float *) malloc(matF_memsize);
    float* h_M = (float *) malloc(matM_memsize);

    float* M_check = (float *) malloc(matM_memsize);

    float* d_A = NULL;
    hipMalloc(&d_A, matA_memsize);
    float* d_F = NULL;
    hipMalloc(&d_F, matF_memsize);
    float* d_M = NULL;
    hipMalloc(&d_M, matM_memsize);

    /*initializing input array*/
    for (int j=0; j < Height; ++j) {
	for (int i=0; i < Width; ++i) {
	    h_A [j*Width + i] = static_cast<float>(j);
	}
    }
    for (int j=0; j < FilterSize; ++j) {
	for (int i=0; i < FilterSize; ++i) {
	    h_F [j*FilterSize + i] = static_cast<float>((j));
	}
    }
    /*correct answer for error checking*/
    for (int row=0; row < Height; ++row) 
    {
	for (int col=0; col < Width; ++col) 
	{
	    float sum = 0.f;
            for(int j =  0; j < FilterSize; ++j) 
	    {
                for(int i = 0; i < FilterSize; ++i) 
	        {
	            int inCol = col + i - FILTER_RADIUS; 		    
	            int inRow = row + j - FILTER_RADIUS;
	            if(inRow >= 0 && inRow < Height && inCol >=0 && inCol < Width) 
	            {
                        sum += h_A[inRow*Width + inCol] * h_F[j*FilterSize + i];     
	            }
	        }
	    }
	    M_check [row*Width + col] = sum;
	}
    }

#ifdef PRINT
    std::cout << "\nWriting A matrix:\n";
    print_matrix(h_A, Width, Height);

    std::cout << "Writing Filter F:\n";
    print_matrix(h_F, FilterSize, FilterSize);

    std::cout << "Writing correct answer for M matrix:\n";
    print_matrix(M_check, Width, Height);
#endif

    hipMemcpy(d_A, h_A, matA_memsize, hipMemcpyHostToDevice);

#ifdef NAIVE
    hipMemcpy(d_F, h_F, matF_memsize, hipMemcpyHostToDevice);
#else  //use constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(F), h_F, matF_memsize);
#endif

//  hipMemset(d_M, 0, matM_memsize);
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;
    hipEventRecord(startEvent, 0);

#ifdef NAIVE
    convolution_naive<<<dimGrid, dimBlock>>>(d_M, d_A, d_F, Height, Width);
#elif CONSTMEM
    convolution_constmem<<<dimGrid, dimBlock>>>(d_M, d_A, Height, Width);
#elif TILED_CONSTMEM_TYPE_1
    convolution_constmem_tiled_type1<<<dimGrid, dimBlock>>>(d_M, d_A, Height, Width);
#elif TILED_CONSTMEM_CACHEHALO_TYPE_1
    convolution_constmem_tiled_cachehalo<<<dimGrid, dimBlock>>>(d_M, d_A, Height, Width);
#endif

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    std::cout << "\nElapsed time to run kernel (ms): " << ms << "\n";

    hipMemcpy(h_M, d_M, matM_memsize, hipMemcpyDeviceToHost); 
  
#ifdef PRINT
    std::cout << "Writing M matrix:\n";
    print_matrix(h_M, Width, Height);
#endif

   check_error(h_M, M_check, Width*Height);

//error_exit:
    /*free memory*/
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    free(h_A);
    free(h_F);
    free(h_M);
    free(M_check);

    hipFree(d_A);
    hipFree(d_F);
    hipFree(d_M);

    hipDeviceReset();
    return 0;
}

